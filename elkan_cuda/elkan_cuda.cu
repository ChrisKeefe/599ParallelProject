#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <stdbool.h>
#include <string.h>
#include <omp.h>

#include "csvparser.h"

#define BLOCKSIZE 1024

using namespace std;

__global__ void init_ubound(int *dev_num_rows, double *dev_u_bounds);
__global__ void ctr_ctr_dist_calc(int *dev_K, int *dev_num_cols, double *dev_ctr_ctr_dists,
                                  double *dev_centers, double *dev_s);
__global__ void elkan(int *dev_num_rows, int *dev_num_cols, double *dev_l_bounds,
                      double *dev_u_bounds, int *dev_clusterings, double *dev_ctr_ctr_dists,
                      double *dev_centers, double *dev_data_matrix, bool *dev_changes, int *dev_K,
                      double *dev_s);
__global__ void reassign(int *dev_num_rows, int *dev_num_cols, int *dev_clusterings, double *dev_cluster_means,
                         double *dev_data_matrix, int *dev_elements_per_cluster);
__global__ void finishReassign(int *dev_num_cols, int *dev_K, double *dev_cluster_means,
                               int *dev_elements_per_cluster);
__global__ void calc_drifts(int *dev_K, int *dev_num_cols, double *dev_centers,
                            double *dev_prev_centers, double *dev_drifts);
__global__ void adjust_bounds(double *dev_u_bounds, double *dev_l_bounds, double *dev_centers,
                              double *dev_prev_centers, int *dev_clusterings, double *dev_drifts,
                              int *dev_num_rows, int *dev_num_cols, int *dev_K);

void warmUpGPU();

void vector_init(double *a, int length) {
  for (int i = 0; i < length; i++) {
    a[i] = 0;
  }
}

void vector_copy(double *dst, double *src, int length) {
  for (int i = 0; i < length; i++) {
    dst[i] = src[i];
  }
}

void vector_add(double *dst, double *a, double *b, int length) {
  for (int i = 0; i < length; i++) {
    dst[i] = a[i] + b[i];
  }
}

void vector_elementwise_avg(double *dst, double *a, int denominator, int length) {
  for (int i = 0; i < length; i++) {
    dst[i] = a[i] / denominator;
  }
}

double vector_L2_norm(double *a, int length) {
  double vec_norm = 0;

  for (int i = 0; i < length; i++) {
    vec_norm += a[i] * a[i];
  }

  return sqrt(vec_norm);
}

void vector_sub(double *dst, double *a, double *b, int length) {
  for (int i = 0; i < length; i++) {
    dst[i] = a[i] - b[i];
  }
}

// Program should take K, a data set (.csv), a delimiter,
// a binary flag data_contains_header, and a binary flag to drop labels
int main(int argc, char *argv[]) {
  // Seed for consistent cluster center selection
  // In a working implementation, seeding would be variable (e.g. time(NULL))
  srand(111);
  CsvParser *reader;
  CsvRow *row;
  int i, j;

  if (argc < 6) {
    printf("Incorrect number of args. Should be 5, received %d\n", argc - 1);
    exit(1);
  }

  int K = atoi(argv[1]);
  char *data_fp = argv[2];
  char *delimiter = argv[3];
  int has_header_row = atoi(argv[4]);
  int drop_labels = atoi(argv[5]);

  // Take in data set
  reader = CsvParser_new(data_fp, delimiter, has_header_row);

  // Get number of columns
  row = CsvParser_getRow(reader);
  int num_cols = CsvParser_getNumFields(row);
  CsvParser_destroy_row(row);

  if (drop_labels){
    num_cols--;
  }

  // Get number of rows like lazy people
  int num_rows = 1;
  while ((row = CsvParser_getRow(reader))){
    num_rows++;
    CsvParser_destroy_row(row);
  }

  // Torch the CsvParser and start again so we can read data in.
  CsvParser_destroy(reader);

  reader = CsvParser_new(data_fp, delimiter, has_header_row);

  double *data_matrix = (double *)malloc(num_rows * num_cols * sizeof(double));

  int row_index = 0;
  while ((row = CsvParser_getRow(reader))){
    const char **row_fields = CsvParser_getFields(row);

    for (int col_index = 0; col_index < num_cols; col_index++) {
      data_matrix[row_index * num_cols + col_index] = atof(row_fields[col_index]);
    }

    CsvParser_destroy_row(row);
    row_index++;
  }

  CsvParser_destroy(reader);

  // Initialize some cluster centers from random rows in our data
  // Given the fact that we will usually have way more rows than centers, we can
  // probably just roll a number and reroll if we already rolled it. Collisions
  // should be relatively infrequent
  double *centers = (double *)malloc(K * num_cols * sizeof(double));
  double *prev_centers = (double *)malloc(K * num_cols * sizeof(double));
  bool collided;

  if (argc == 7) {
    int center_indices[3] = {12, 67, 106};
    for (i = 0; i < K; i ++) {
      vector_copy(centers + i * num_cols, data_matrix + center_indices[i] * num_cols, num_cols);
    }
  } else {
    for (i = 0; i < K; i++) {
      int center_indices[K];
      collided = true;

      while (collided) {
        center_indices[i] = rand() % num_rows;
        collided = false;

        for (j = 0; j < i; j++) {
          if (center_indices[j] == center_indices[i]) {
            collided = true;
            break;
          }
        }

        vector_copy(centers + i * num_cols, data_matrix + center_indices[i] * num_cols, num_cols);
      }
    }
  }

  printf("Initial cluster centers:\n");
  for (i = 0; i < K; i++) {
    for (j = 0; j < num_cols; j++) {
      printf("%f ", centers[i * num_cols + j]);
    }
    printf("\n");
  }
  printf("\n");

  // Create vars and allocate data for GPU
  const unsigned int totalBlocks = ceil(num_rows * 1.0 / BLOCKSIZE);
  warmUpGPU();

  int num_iterations = 0;
  int *clusterings = (int *)calloc(num_rows, sizeof(int));
  double *cluster_means = (double *)malloc(num_cols * K * sizeof(double));
  int elements_per_cluster[K];
  bool changes;

  double *l_bounds = (double *)calloc(num_rows * K, sizeof(double));
  double *u_bounds = (double *)calloc(num_rows, sizeof(double));
  double *ctr_ctr_dists = (double *)malloc(K * K * sizeof(double));
  double *drifts = (double *)malloc(K * sizeof(double));

  // These need better names
  double s[K];

  int this_ctr, this_pt;
  double tmp_diff[num_cols];
  double min_diff;

  double *dev_data_matrix;
  double *dev_centers;
  double *dev_prev_centers;
  double *dev_cluster_means;
  double *dev_u_bounds;
  double *dev_l_bounds;
  double *dev_drifts;
  double *dev_s;
  double *dev_z;
  double *dev_ctr_ctr_dists;
  int *dev_elements_per_cluster;
  int *dev_clusterings;
  bool *dev_changes;
  int *dev_num_rows;
  int *dev_num_cols;
  int *dev_K;

  double kernel_start;
  double kernel_time = 0;
  double transfer_time = 0;
  double t_cpu_start = 0;
  double cpu_time;

  hipError_t errCode = hipSuccess;

  double t_start = omp_get_wtime();
  double t_transfer_start = t_start;

  errCode = hipMalloc(&dev_data_matrix, sizeof(double) * num_rows * num_cols);
  if (errCode != hipSuccess) {
    cout << "\nError: data_matrix alloc error with code " << errCode << endl;
  }

  errCode = hipMemcpy(dev_data_matrix, data_matrix, sizeof(double) * num_rows * num_cols, hipMemcpyHostToDevice);
  if (errCode != hipSuccess) {
    cout << "\nError: data_matrix memcpy error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_elements_per_cluster, sizeof(int) * K);
  if (errCode != hipSuccess) {
    cout << "\nError: elements per cluster alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_cluster_means, sizeof(double) * num_cols * K);
  if (errCode != hipSuccess) {
    cout << "\nError: cluster means alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_centers, sizeof(double) * K * num_cols);
  if (errCode != hipSuccess) {
    cout << "\nError: centers alloc error with code " << errCode << endl;
  }

  errCode = hipMemcpy(dev_centers, centers, sizeof(double) * K * num_cols, hipMemcpyHostToDevice);
  if (errCode != hipSuccess) {
    cout << "\nError: centers memcpy error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_clusterings, sizeof(int) * num_rows);
  if (errCode != hipSuccess) {
    cout << "\nError: clusterings alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_changes, sizeof(bool));
  if (errCode != hipSuccess) {
    cout << "\nError: changes alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_num_rows, sizeof(int));
  if (errCode != hipSuccess) {
    cout << "\nError: num_rows alloc error with code " << errCode << endl;
  }

  errCode = hipMemcpy(dev_num_rows, &num_rows, sizeof(int), hipMemcpyHostToDevice);
  if (errCode != hipSuccess) {
    cout << "\nError: num_rows memcpy error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_num_cols, sizeof(int));
  if (errCode != hipSuccess) {
    cout << "\nError: num_cols alloc error with code " << errCode << endl;
  }

  errCode = hipMemcpy(dev_num_cols, &num_cols, sizeof(int), hipMemcpyHostToDevice);
  if (errCode != hipSuccess) {
    cout << "\nError: num_cols memcpy error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_K, sizeof(int));
  if (errCode != hipSuccess) {
    cout << "\nError: K alloc error with code " << errCode << endl;
  }

  errCode = hipMemcpy(dev_K, &K, sizeof(int), hipMemcpyHostToDevice);
  if (errCode != hipSuccess) {
    cout << "\nError: K memcpy error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_u_bounds, sizeof(double) * num_rows);
  if (errCode != hipSuccess) {
    cout << "\nError: u bounds alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_l_bounds, sizeof(double) * num_rows * K);
  if (errCode != hipSuccess) {
    cout << "\nError: l bounds alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_drifts, sizeof(double) * K);
  if (errCode != hipSuccess) {
    cout << "\nError: drifts alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_prev_centers, sizeof(double) * K * num_cols);
  if (errCode != hipSuccess) {
    cout << "\nError: prev centers alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_s, sizeof(double) * K);
  if (errCode != hipSuccess) {
    cout << "\nError: s alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_z, sizeof(double));
  if (errCode != hipSuccess) {
    cout << "\nError: z alloc error with code " << errCode << endl;
  }

  errCode = hipMalloc(&dev_ctr_ctr_dists, sizeof(double) * K * K);
  if (errCode != hipSuccess) {
    cout << "\nError: ctr ctr dists alloc error with code " << errCode << endl;
  }

  errCode = hipMemset(dev_clusterings, 0, num_rows * sizeof(int));
  if (errCode != hipSuccess) {
    cout << "\nError: memsetting cluster means error with code " << errCode << endl;
  }

  transfer_time += omp_get_wtime() - t_transfer_start;
  cout << "Initial transfer time: " << transfer_time << " seconds" << endl;

  // #########################
  // # BEGIN ELKAN MAIN LOOP #
  // #########################

  // TODO: I suspect we're going to need additional memory allocations:
  // u_bound, l_bound, s, z, drifts, ctr_ctr_dists, prev_clousterings, bound_not_tight?

  // #pragma omp parallel for private(this_pt) shared(num_rows, u_bounds)
  // for (this_pt = 0; this_pt < num_rows; this_pt++) {
  //   u_bounds[this_pt] = INFINITY;
  // }

  kernel_start = omp_get_wtime();
  init_ubound<<<totalBlocks, BLOCKSIZE>>>(dev_num_rows, dev_u_bounds);
  kernel_time += omp_get_wtime() - kernel_start;

  while (1) {
    changes = false;
    // send changes flag to GPU and time the transfer

    // ###############################################################################
    // Calculate center-center distances with OpenMP (K>=64 uncommon, xfer too costly)
    // ###############################################################################
    // t_cpu_start = omp_get_wtime();
    // #pragma omp parallel for private (i, j, tmp_diff, min_diff) \
    //     shared(ctr_ctr_dists, centers, num_cols)
    // for (i = 0; i < K; i++) {
    //   min_diff = INFINITY;

    //   for (j = 0; j < K; j++) {
    //     vector_sub(tmp_diff, centers + i * num_cols, centers + j * num_cols, num_cols);
    //     ctr_ctr_dists[i * K + j] = vector_L2_norm(tmp_diff, num_cols);

    //     if (ctr_ctr_dists[i * K + j] < min_diff && i != j) {
    //       min_diff = ctr_ctr_dists[i * K + j];
    //     }
    //   }

    //   s[i] = min_diff / 2;
    // }
    // cpu_time += omp_get_wtime() - t_cpu_start;
    t_transfer_start = omp_get_wtime();
    errCode = hipMemcpy(dev_changes, &changes, sizeof(bool), hipMemcpyHostToDevice);
    if (errCode != hipSuccess) {
      cout << "\nError: changes memcpy error with code " << errCode << endl;
    }

    // errCode = hipMemcpy(dev_s, s, sizeof(double) * K, hipMemcpyHostToDevice);
    // if (errCode != hipSuccess) {
    //   cout << "\nError: s memcpy error with code " << errCode << endl;
    // }

    // errCode = hipMemcpy(dev_ctr_ctr_dists, ctr_ctr_dists, sizeof(double) * K * K, hipMemcpyHostToDevice);
    // if (errCode != hipSuccess) {
    //   cout << "\nError: ctr ctr dists memcpy error with code " << errCode << endl;
    // }

    // errCode = hipMemcpy(dev_u_bounds, u_bounds, sizeof(double) * num_rows, hipMemcpyHostToDevice);
    // if (errCode != hipSuccess) {
    //   cout << "\nError: u_bounds memcpy error with code " << errCode << endl;
    // }

    // errCode = hipMemcpy(dev_l_bounds, l_bounds, sizeof(double) * num_rows * K, hipMemcpyHostToDevice);
    // if (errCode != hipSuccess) {
    //   cout << "\nError: l_bounds memcpy error with code " << errCode << endl;
    // }
    transfer_time += omp_get_wtime() - t_transfer_start;

    // #################################
    // Assign points to cluster centers
    // #################################
    // TODO: transfer data, implement and run assign_points kernel, time
    // Assign points to cluster centers
    kernel_start = omp_get_wtime();
    ctr_ctr_dist_calc<<<totalBlocks, BLOCKSIZE>>>(dev_K, dev_num_cols, dev_ctr_ctr_dists, dev_centers, dev_s);
    hipDeviceSynchronize();
    elkan<<<totalBlocks, BLOCKSIZE>>>(dev_num_rows, dev_num_cols, dev_l_bounds, dev_u_bounds,
                                      dev_clusterings, dev_ctr_ctr_dists, dev_centers, dev_data_matrix,
                                      dev_changes, dev_K, dev_s);
    hipDeviceSynchronize();
    kernel_time += omp_get_wtime() - kernel_start;

    // ######################################################################
    // If we didn't change any cluster assignments, we've reached convergence
    // ######################################################################
    errCode = hipMemcpy(&changes, dev_changes, sizeof(bool), hipMemcpyDeviceToHost);
    if (errCode != hipSuccess) {
      cout << "\nError: getting changes from GPU error with code " << errCode << endl;
    }

    if (!changes) {
      break;
    }

    num_iterations++;

    // Capture current centers for later re-use
    // memcpy(prev_centers, centers, num_cols * K * sizeof(double));

    // #######################################
    // Find cluster means and reassign centers
    // #######################################
    t_transfer_start = omp_get_wtime();
    errCode = hipMemset(dev_elements_per_cluster, 0, K * sizeof(int));
    if (errCode != hipSuccess) {
      cout << "\nError: memsetting elements per cluster error with code " << errCode << endl;
    }

    errCode = hipMemset(dev_cluster_means, 0, num_cols * K * sizeof(double));
    if (errCode != hipSuccess) {
      cout << "\nError: memsetting cluster means error with code " << errCode << endl;
    }

    errCode = hipMemcpy(dev_prev_centers, dev_centers, sizeof(double) * K * num_cols, hipMemcpyDeviceToDevice);
    if (errCode != hipSuccess) {
      cout << "\nError: centers to prev centers memcpy error with code " << errCode << endl;
    }
    transfer_time += omp_get_wtime() - t_transfer_start;

    kernel_start = omp_get_wtime();
    reassign<<<totalBlocks, BLOCKSIZE>>>(dev_num_rows, dev_num_cols, dev_clusterings, dev_cluster_means, dev_data_matrix, dev_elements_per_cluster);
    hipDeviceSynchronize();

    finishReassign<<<totalBlocks, BLOCKSIZE>>>(dev_num_cols, dev_K, dev_cluster_means, dev_elements_per_cluster);
    hipDeviceSynchronize();
    kernel_time += omp_get_wtime() - kernel_start;

    // Replace the old cluster means with the new using only three assignments.
    double *temp = dev_centers;
    dev_centers = dev_cluster_means;
    dev_cluster_means = temp;

    // ###########################################
    // Compute centroid drift since last iteration
    // ###########################################
    // #pragma omp parallel for private(this_ctr, tmp_diff) \
    //         shared(centers, prev_centers, num_cols, drifts)
    // for (this_ctr = 0; this_ctr < K; this_ctr++) {
    //   vector_sub(tmp_diff, &centers[this_ctr], &prev_centers[this_ctr], num_cols);
    //   drifts[this_ctr] = vector_L2_norm(tmp_diff, num_cols);
    // }
    // cpu_time += omp_get_wtime() - t_cpu_start;

    kernel_start = omp_get_wtime();
    calc_drifts<<<totalBlocks, BLOCKSIZE>>>(dev_K, dev_num_cols, dev_centers,
                                            dev_prev_centers, dev_drifts);
    hipDeviceSynchronize();
    // ###########################################
    // Adjust bounds to account for centroid drift
    // ###########################################
    // TODO: transfer data, call adjust_bounds (below), time
    // errCode = hipMemcpy(dev_drifts, drifts, sizeof(double) * K, hipMemcpyHostToDevice);
    // if (errCode != hipSuccess) {
    //   cout << "\nError: drifts memcpy error with code " << errCode << endl;
    // }

    // errCode = hipMemcpy(dev_centers, centers, sizeof(double) * K * num_cols, hipMemcpyHostToDevice);
    // if (errCode != hipSuccess) {
    //   cout << "\nError: centers memcpy error with code " << errCode << endl;
    // }

    // errCode = hipMemcpy(dev_prev_centers, prev_centers, sizeof(double) * K * num_cols, hipMemcpyHostToDevice);
    // if (errCode != hipSuccess) {
    //   cout << "\nError: prev centers memcpy error with code " << errCode << endl;
    // }
    adjust_bounds<<<totalBlocks, BLOCKSIZE>>>(dev_u_bounds, dev_l_bounds, dev_centers,
                                              dev_prev_centers, dev_clusterings, dev_drifts,
                                              dev_num_rows, dev_num_cols, dev_K);
    hipDeviceSynchronize();
    kernel_time += omp_get_wtime() - kernel_start;
  }

  t_transfer_start = omp_get_wtime();
  errCode = hipMemcpy(centers, dev_centers, sizeof(double) * K * num_cols, hipMemcpyDeviceToHost);
  if (errCode != hipSuccess) {
    cout << "\nError: getting centers from GPU error with code " << errCode << endl;
  }
  double tend = omp_get_wtime();
  transfer_time += tend - t_transfer_start;

  printf("\nFinal cluster centers:\n");
  for (i = 0; i < K; i++) {
    for (j = 0; j < num_cols; j++) {
      printf("%f ", centers[i * num_cols + j]);
    }
    printf("\n");
  }

  printf("\nNum iterations: %d\n", num_iterations);
  printf("Time taken for %d clusters: %f seconds\nkernel: %f seconds"
         "\ntotaltransfer: %f seconds\nCPU time: %f seconds\n\n",
         K, tend - t_start, kernel_time, transfer_time, cpu_time);

  free(data_matrix);
  free(clusterings);

  exit(0);
}


__global__ void init_ubound(int *dev_num_rows, double *dev_u_bounds) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= *dev_num_rows) {
    return;
  }

  dev_u_bounds[tid] = INFINITY;
}


__global__ void ctr_ctr_dist_calc(int *dev_K, int *dev_num_cols, double *dev_ctr_ctr_dists,
                                  double *dev_centers, double *dev_s) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= *dev_K) {
    return;
  }

  double min_diff = INFINITY;
  double vec_norm = 0;
  double temp;

  int j;
  for (int i = 0; i < *dev_K; i++) {
    if (i == tid) {
      dev_ctr_ctr_dists[tid * *dev_K + i] = 0;
      continue;
    }

    // vector_sub(tmp_diff, centers + i * num_cols, centers + j * num_cols, num_cols);
    // ctr_ctr_dists[i * K + j] = vector_L2_norm(tmp_diff, num_cols);

    for (j = 0; j < *dev_num_cols; j++) {
      temp = dev_centers[tid * *dev_num_cols + j] -
             dev_centers[i * *dev_num_cols + j];
      vec_norm += temp * temp;
    }
    dev_ctr_ctr_dists[tid * *dev_K + i] = sqrt(vec_norm);

    if (dev_ctr_ctr_dists[tid * *dev_K + i] < min_diff) {
      min_diff = dev_ctr_ctr_dists[tid * *dev_K + i];
    }
  }

  dev_s[tid] = min_diff / 2;
}


__global__ void elkan(int *dev_num_rows, int *dev_num_cols, double *dev_l_bounds,
                      double *dev_u_bounds, int *dev_clusterings, double *dev_ctr_ctr_dists,
                      double *dev_centers, double *dev_data_matrix, bool *dev_changes, int *dev_K,
                      double *dev_s) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= *dev_num_rows) {
    return;
  }

  double z;
  bool ubound_not_tight;
  double temp;
  double vec_norm;
  int i = 0;

  if (dev_u_bounds[tid] > dev_s[dev_clusterings[tid]]) {
    ubound_not_tight = true;

    for(int this_ctr = 0; this_ctr < *dev_K; this_ctr++) {
      z = max(dev_l_bounds[tid * *dev_K + this_ctr],
              dev_ctr_ctr_dists[dev_clusterings[tid] * *dev_K + this_ctr] / 2);

      if (this_ctr == dev_clusterings[tid] || dev_u_bounds[tid] <= z) {
        continue;
      }

      if (ubound_not_tight) {
        vec_norm = 0;
        for (i = 0; i < *dev_num_cols; i++) {
          temp = dev_data_matrix[tid * *dev_num_cols + i] -
                 dev_centers[dev_clusterings[tid] * *dev_num_cols + i];
          vec_norm += temp * temp;
        }
        dev_u_bounds[tid] = sqrt(vec_norm);
        ubound_not_tight = false;

        if (dev_u_bounds[tid] <= z) {
          continue;
        }
      }

      vec_norm = 0;
      for (i = 0; i < *dev_num_cols; i++) {
        temp = dev_data_matrix[tid * *dev_num_cols + i] -
               dev_centers[this_ctr * *dev_num_cols + i];
        vec_norm += temp * temp;
      }
      dev_l_bounds[tid * *dev_K + this_ctr] = sqrt(vec_norm);
      if(dev_l_bounds[tid * *dev_K + this_ctr] < dev_u_bounds[tid]) {
        // NOTE: There is an acceptable data race on changes. Threads only ever
        // set it to true; lost updates are inconsequential. No need to slow
        // things down for safety.
        *dev_changes = true;
        dev_clusterings[tid] = this_ctr;
        dev_u_bounds[tid] = dev_l_bounds[tid * *dev_K + this_ctr];
      }
    }
  }
}


/*
Reassigns centroids to their new cluster means
*/
__global__ void reassign(int *dev_num_rows, int *dev_num_cols, int *dev_clusterings, double *dev_cluster_means,
                         double *dev_data_matrix, int *dev_elements_per_cluster) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= *dev_num_rows) {
    return;
  }

  unsigned int cluster = dev_clusterings[tid];

  for (unsigned int i = 0; i < *dev_num_cols; i++) {
    atomicAdd(&dev_cluster_means[cluster * *dev_num_cols + i], dev_data_matrix[tid * *dev_num_cols + i]);
  }

  atomicAdd(&dev_elements_per_cluster[cluster], int(1));
}


__global__ void finishReassign(int *dev_num_cols, int *dev_K, double *dev_cluster_means,
                               int *dev_elements_per_cluster) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= *dev_K) {
    return;
  }

  for (int i = 0; i < *dev_num_cols; i++) {
    dev_cluster_means[tid * *dev_num_cols + i] /= dev_elements_per_cluster[tid];
  }
}


__global__ void calc_drifts(int *dev_K, int *dev_num_cols, double *dev_centers,
                            double *dev_prev_centers, double *dev_drifts) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= *dev_K) {
    return;
  }

  double vec_norm = 0;
  double temp;

  for (int i = 0; i < *dev_num_cols; i++) {
    temp = dev_centers[tid * *dev_num_cols + i] -
           dev_prev_centers[tid * *dev_num_cols + i];
    vec_norm += temp * temp;
  }
  dev_drifts[tid] = sqrt(vec_norm);
}


/*
Adjusts the upper and lower bounds to accomodate for centroid drift
*/
__global__ void adjust_bounds(double *dev_u_bounds, double *dev_l_bounds, double *dev_centers,
                              double *dev_prev_centers, int *dev_clusterings, double *dev_drifts,
                              int *dev_num_rows, int *dev_num_cols, int *dev_K) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid >= *dev_num_rows) return;

  double temp;
  double vec_norm = 0;
  for (int i = 0; i < *dev_num_cols; i++) {
    temp = dev_centers[dev_clusterings[tid]] - dev_prev_centers[dev_clusterings[tid]];
    vec_norm += temp * temp;
  }
  dev_u_bounds[tid] += sqrt(vec_norm);

  for (int this_ctr = 0; this_ctr < *dev_K; this_ctr++) {
    dev_l_bounds[tid * (*dev_K) + this_ctr] -= dev_drifts[this_ctr];
  }
}


/*
Warms up the GPU so that timings are accurate/consistent
*/
void warmUpGPU() {
  hipDeviceSynchronize();
  return;
}
